#include "hip/hip_runtime.h"
#include "DynamicFusionProcessor.h"
#include "TsdfVolume.h"
#include "WarpField.h"
#include "device_utils.h"
#include <hip/hip_vector_types.h>
namespace dfusion
{
	__device__ __forceinline__ int sign(float a)
	{
		return (a > 0) - (a < 0);
	}

	template<int knnNotZero>
	__device__ __forceinline__ static Tbx::Dual_quat_cu calc_dual_quat_blend_on_voxel(
		hipTextureObject_t knnTex, hipTextureObject_t nodesDqVwTex, float2 tsdf_prev,
		int x, int y, int z, float3 origion, float voxelSize, float inv_dw_for_fusion2,
		float nodeRadius, float marchingCube_weightThre, float& fusion_weight, bool& suc)
	{
		Tbx::Dual_quat_cu dq_blend(Tbx::Quat_cu(0, 0, 0, 0), Tbx::Quat_cu(0, 0, 0, 0));
		fusion_weight = 0.f;
		suc = false;

		// 
		float3 p = make_float3(x*voxelSize, y*voxelSize, z*voxelSize) + origion;
		KnnIdx knnIdx = read_knn_tex(knnTex, x, y, z);

		// the first quat
		float4 q0, q1, vw;
		int nn3;
		//Tbx::Dual_quat_cu dq_avg;
		nn3 = knn_k(knnIdx, 0) * 3;
		tex1Dfetch(&q0, nodesDqVwTex, nn3 + 0);
		tex1Dfetch(&q1, nodesDqVwTex, nn3 + 1);
		tex1Dfetch(&vw, nodesDqVwTex, nn3 + 2);
		float dist2_0 = norm2(make_float3(vw.x - p.x, vw.y - p.y, vw.z - p.z));
		float w_0 = __expf(-dist2_0 * 0.5f * inv_dw_for_fusion2);

		// ldp hack here: if the voxel is too far from existed node, we just 
		// assume its transformation to be identity and fuse it
		// this helps to preserve empty place.
		//if (w_0 < Tbx::Dual_quat_cu::epsilon())
		//{
		//	suc = true;
		//	return Tbx::Dual_quat_cu::identity();
		//}

		dq_blend = pack_dual_quat(q0, q1);
		fusion_weight += sqrt(dist2_0);

		// the other quats
		int k = 1;
#pragma unroll
		for (; k < KnnK; k++)
		{
			if (knn_k(knnIdx, k) >= WarpField::MaxNodeNum)
				break;
			nn3 = knn_k(knnIdx, k) * 3;
			tex1Dfetch(&q0, nodesDqVwTex, nn3 + 0);
			tex1Dfetch(&q1, nodesDqVwTex, nn3 + 1);
			tex1Dfetch(&vw, nodesDqVwTex, nn3 + 2);
			Tbx::Dual_quat_cu dq = pack_dual_quat(q0, q1);

			// note: we store 1.f/radius in vw.w
			float dist2 = norm2(make_float3(vw.x - p.x, vw.y - p.y, vw.z - p.z));
			float w = __expf(-(dist2 - dist2_0) * 0.5f * inv_dw_for_fusion2)
				 *sign(dq_blend[0] * dq[0] + dq_blend[1] * dq[1] + dq_blend[2] * dq[2] + dq_blend[3] * dq[3]);
			dq_blend += dq*w;
			fusion_weight += sqrt(dist2);
		}
		dq_blend *= 1.f/dq_blend.norm();
		fusion_weight = float(k) * nodeRadius / fusion_weight;
		suc = true;
		return dq_blend;
	}

	template<>
	__device__ __forceinline__ static Tbx::Dual_quat_cu calc_dual_quat_blend_on_voxel<0>(
		hipTextureObject_t knnTex, hipTextureObject_t nodesDqVwTex, float2 tsdf_prev,
		int x, int y, int z, float3 origion, float voxelSize, float inv_dw_for_fusion2,
		float nodeRadius, float marchingCube_weightThre, float& fusion_weight, bool& suc)
	{
		fusion_weight = marchingCube_weightThre + 1e-5f;
		suc = true;
		return Tbx::Dual_quat_cu::identity();
	}


	texture<depthtype, hipTextureType2D, hipReadModeElementType> g_depth_tex;
	texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat> g_color_tex;

	struct Fusioner
	{
		PtrStepSz<depthtype> depth;

		hipSurfaceObject_t volumeTex;
		int3 volume_resolution;
		float3 origion;
		float nodeRadius;
		float voxel_size;
		float tranc_dist;
		float max_weight;
		Intr intr;
		float inv_dw_for_fusion2;
		float marchingCube_weightThre;

		hipTextureObject_t knnTex;
		hipTextureObject_t nodesDqVwTex;
		Tbx::Mat3 Rv2c;
		Tbx::Point3 tv2c;

		template<int maxK>
		__device__ __forceinline__ void fusion(int x, int y, int z)
		{
			TsdfData rawTsdf = read_tsdf_surface(volumeTex, x, y, z);
			float2 tsdf_weight_prev = unpack_tsdf(rawTsdf);
			float fusion_weight = 0;
			bool suc = true;
			Tbx::Dual_quat_cu dq = calc_dual_quat_blend_on_voxel<maxK>(
				knnTex, nodesDqVwTex, tsdf_weight_prev, x, y, z, origion, voxel_size, inv_dw_for_fusion2,
				nodeRadius, marchingCube_weightThre, fusion_weight, suc);

			if (!suc)
				return;

			float3 cxyz = convert(Rv2c*(dq.transform(Tbx::Point3(x*voxel_size + origion.x,
				y*voxel_size+origion.y, z*voxel_size+origion.z))) + tv2c);

			float3 uvd = intr.xyz2uvd(cxyz);
			int2 coo = make_int2(__float2int_rn(uvd.x), __float2int_rn(uvd.y));

			if (uvd.x >= 0 && uvd.x < depth.cols && uvd.y >= 0 && uvd.y < depth.rows)
			{
				float depthVal = tex2D(g_depth_tex, coo.x, coo.y)*0.001f;
				float3 dxyz = intr.uvd2xyz(make_float3(coo.x, coo.y, depthVal));
				float sdf = cxyz.z - dxyz.z;

				if (depthVal > KINECT_NEAREST_METER && sdf >= -tranc_dist)
				{
					float tsdf = min(1.0f, sdf / tranc_dist);
					float tsdf_new = (tsdf_weight_prev.x * tsdf_weight_prev.y + fusion_weight * tsdf)
						/ (tsdf_weight_prev.y + fusion_weight);
					float weight_new = min(tsdf_weight_prev.y + fusion_weight, max_weight);
					float4 color = make_float4(0, 0, 0, 0);
#ifdef ENABLE_COLOR_FUSION
					float4 newColor = tex2D(g_color_tex, coo.x, coo.y);
					if (newColor.x != 0.f && newColor.y != 0.f
						&& newColor.z != 0.f)
					{
#if 0
						color = unpack_tsdf_rgba(rawTsdf) * 0.0f +
							tex2D(g_color_tex, coo.x, coo.y) * 1.f;
#else
						color = (unpack_tsdf_rgba(rawTsdf) * tsdf_weight_prev.y +
							fusion_weight * newColor)
							/ (tsdf_weight_prev.y + fusion_weight);
#endif
					}
#endif
					write_tsdf_surface(volumeTex, pack_tsdf(tsdf_new, weight_new,
						color), x, y, z);
				}
			}
		}
	};

	template<int maxK>
	__global__ void tsdf23( Fusioner fs)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;
		int z = threadIdx.z + blockIdx.z * blockDim.z;

		if (x >= fs.volume_resolution.x || y >= fs.volume_resolution.y || z >= fs.volume_resolution.z)
			return;

		fs.fusion<maxK>(x, y, z);
	}// __global__

	void DynamicFusionProcessor::fusion()
	{
		dim3 block(32, 8, 2);
		dim3 grid(divUp(m_volume->getResolution().x, block.x), 
			divUp(m_volume->getResolution().y, block.y),
			divUp(m_volume->getResolution().z, block.z));

		// bind src to texture
		g_depth_tex.filterMode = hipFilterModePoint;
		size_t offset;
		hipChannelFormatDesc desc = hipCreateChannelDesc<depthtype>();
		hipBindTexture2D(&offset, &g_depth_tex, m_depth_input.ptr(), &desc, 
			m_depth_input.cols(), m_depth_input.rows(), m_depth_input.step());
		assert(offset == 0);
#ifdef ENABLE_COLOR_FUSION
		g_color_tex.filterMode = hipFilterModePoint;
		desc = hipCreateChannelDesc<uchar4>();
		hipBindTexture2D(&offset, &g_color_tex, m_color_input.ptr(), &desc,
			m_color_input.cols(), m_color_input.rows(), m_color_input.step());
		assert(offset == 0);
#endif

		Fusioner fs;
		fs.depth = m_depth_input;
		fs.volumeTex = m_volume->getSurface();
		fs.volume_resolution = m_volume->getResolution();
		fs.origion = m_volume->getOrigion();
		fs.nodeRadius = m_param.warp_radius_search_epsilon;
		fs.voxel_size = m_volume->getVoxelSize();
		fs.tranc_dist = m_volume->getTsdfTruncDist();
		fs.max_weight = m_param.fusion_max_weight;
		fs.intr = m_kinect_intr;
		fs.inv_dw_for_fusion2 = 1.f / (m_param.warp_param_dw_for_fusion*m_param.warp_param_dw_for_fusion);
		fs.marchingCube_weightThre = m_param.marchingCube_min_valied_weight;

		fs.knnTex = m_warpField->getKnnFieldTexture();
		fs.nodesDqVwTex = m_warpField->getNodesDqVwTexture();	
		Tbx::Transfo tr = m_warpField->get_rigidTransform();
		fs.Rv2c = tr.get_mat3();
		fs.tv2c = Tbx::Point3(tr.get_translation());

		int maxK = min(KnnK, m_warpField->getNumNodesInLevel(0));

		if (maxK == 0)
			tsdf23<0> << <grid, block >> >(fs);
		else
			tsdf23<1> << <grid, block >> >(fs);

		hipUnbindTexture(&g_depth_tex);
#ifdef ENABLE_COLOR_FUSION
		hipUnbindTexture(&g_color_tex);
#endif

		cudaSafeCall(hipGetLastError(), "DynamicFusionProcessor::fusion()");
	}

#pragma region --min-filter

	const static int BLOCK_DIM_X = 32;
	const static int BLOCK_DIM_Y = 16;
	const static int MAX_FILTER_RADIUS = 16;
	const static int X_HALO_STEPS = (MAX_FILTER_RADIUS + BLOCK_DIM_X - 1) / BLOCK_DIM_X;
	const static int Y_HALO_STEPS = (MAX_FILTER_RADIUS + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y;
	const static int X_PATCH_PER_BLOCK = 4;
	const static int Y_PATCH_PER_BLOCK = 4;

	template<int Radius>
	__global__ void erose_filter_row(uchar4*  __restrict__ dst,
		const uchar4*  __restrict__ src, int nX, int nY, int pitch)
	{
		// Data cache: threadIdx.x , threadIdx.y
		enum{ SMEM_X_LEN = (X_PATCH_PER_BLOCK + 2 * X_HALO_STEPS) * BLOCK_DIM_X };
		enum{ SMEM_Y_LEN = BLOCK_DIM_Y };
		__shared__ uchar4 smem[SMEM_Y_LEN][SMEM_X_LEN];

		const int baseX = (blockIdx.x * X_PATCH_PER_BLOCK - X_HALO_STEPS) * BLOCK_DIM_X + threadIdx.x;
		const int baseY = blockIdx.y * BLOCK_DIM_Y + threadIdx.y;
		if (baseY >= nY)
			return;

		src += baseY * pitch + baseX;
		dst += baseY * pitch + baseX;

		//Load main data and right halo
#pragma unroll
		for (int patchId = 0; patchId < X_HALO_STEPS * 2 + X_PATCH_PER_BLOCK; patchId++)
		{
			const int pbx = patchId * BLOCK_DIM_X;
			smem[threadIdx.y][threadIdx.x + pbx] =
				(pbx + baseX < nX && pbx + baseX >= 0) ? src[pbx] : make_uchar4(255,255,255,255);
		}

		//Compute and store results
		__syncthreads();
#pragma unroll
		for (int patchId = X_HALO_STEPS; patchId < X_HALO_STEPS + X_PATCH_PER_BLOCK; patchId++)
		{
			const int pbx = patchId * BLOCK_DIM_X;
			if (baseX + pbx < nX)
			{
				uchar4 s = smem[threadIdx.y][threadIdx.x + pbx];
#pragma unroll
				for (int j = -Radius; j <= Radius; j++)
				{
					if (smem[threadIdx.y][threadIdx.x + pbx + j].x == 0
						&& smem[threadIdx.y][threadIdx.x + pbx + j].y == 0
						&& smem[threadIdx.y][threadIdx.x + pbx + j].z == 0)
						s = make_uchar4(0, 0, 0, 0);
				}
				dst[pbx] = s;
			}
		}
	}

	template<int Radius>
	__global__ void erose_filter_col(uchar4*  __restrict__ dst,
		const uchar4*  __restrict__ src, int nX, int nY, int pitch)
	{
		// Data cache: threadIdx.x , threadIdx.y
		enum{ SMEM_X_LEN = BLOCK_DIM_X };
		enum{ SMEM_Y_LEN = (Y_PATCH_PER_BLOCK + 2 * Y_HALO_STEPS) * BLOCK_DIM_Y };
		__shared__ uchar4 smem[SMEM_Y_LEN][SMEM_X_LEN];

		const int baseX = blockIdx.x * BLOCK_DIM_X + threadIdx.x;
		const int baseY = (blockIdx.y * Y_PATCH_PER_BLOCK - Y_HALO_STEPS) * BLOCK_DIM_Y + threadIdx.y;
		if (baseX >= nX)
			return;

		src += baseY * pitch + baseX;
		dst += baseY * pitch + baseX;

		//Load main data and lower halo
#pragma unroll
		for (int patchId = 0; patchId < Y_HALO_STEPS * 2 + Y_PATCH_PER_BLOCK; patchId++)
		{
			const int pby = patchId * BLOCK_DIM_Y;
			smem[threadIdx.y + pby][threadIdx.x] =
				(pby + baseY < nY && pby + baseY >= 0) ? src[pby * pitch] : make_uchar4(255,255,255,255);
		}

		//Compute and store results
		__syncthreads();
#pragma unroll
		for (int patchId = Y_HALO_STEPS; patchId < Y_HALO_STEPS + Y_PATCH_PER_BLOCK; patchId++)
		{
			const int pby = patchId * BLOCK_DIM_Y;
			if (baseY + pby < nY)
			{
				uchar4 s = smem[threadIdx.y + pby][threadIdx.x];
#pragma unroll
				for (int j = -Radius; j <= Radius; j++)
				{
					if (smem[threadIdx.y + pby + j][threadIdx.x].x == 0
						&& smem[threadIdx.y + pby + j][threadIdx.x].y == 0
						&& smem[threadIdx.y + pby + j][threadIdx.x].z == 0)
						s = make_uchar4(0, 0, 0, 0);
				}
				dst[pby * pitch] = s;
			}
		}
	}


	template<int Radius>
	static void erose_filter_row_caller(uchar4* dst, const uchar4* src,
		int nX, int nY, int pitch)
	{
		dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
		dim3 grid(divUp(nX, block.x*X_PATCH_PER_BLOCK), divUp(nY, block.y), 1);

		erose_filter_row<Radius> << <grid, block >> >(dst, src, nX, nY, pitch);
	}

	template<int Radius>
	static void erose_filter_col_caller(uchar4* dst, const uchar4* src,
		int nX, int nY, int pitch)
	{
		dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
		dim3 grid(divUp(nX, block.x), divUp(nY, block.y*Y_PATCH_PER_BLOCK), 1);

		erose_filter_col<Radius> << <grid, block >> >(dst, src, nX, nY, pitch);
	}

	void erose_filter(uchar4* dst_d, const uchar4* src_d,
		int nX, int nY, int pitch,
		int radius, int dim)
	{
		if (src_d == dst_d)
			throw std::exception("min_filter: src and dst cannot be the same memory!");
		if (radius <= 0 || radius >= MAX_FILTER_RADIUS)
			throw std::exception("min_filter: error, non supported kernel size!");
		if (dim > 2 || dim < 0)
			throw std::exception("min_filter: illegal input dim");

		typedef void(*row_caller_t)(uchar4* dst, const uchar4* src,
			int nX, int nY, int pitch);
		typedef void(*col_caller_t)(uchar4* dst, const uchar4* src,
			int nX, int nY, int pitch);
		static const row_caller_t row_callers[MAX_FILTER_RADIUS] =
		{
			0, erose_filter_row_caller<1>, erose_filter_row_caller<2>, erose_filter_row_caller<3>,
			erose_filter_row_caller<4>, erose_filter_row_caller<5>, erose_filter_row_caller<6>,
			erose_filter_row_caller<7>, erose_filter_row_caller<8>, erose_filter_row_caller<9>,
			erose_filter_row_caller<10>, erose_filter_row_caller<11>, erose_filter_row_caller<12>,
			erose_filter_row_caller<13>, erose_filter_row_caller<14>, erose_filter_row_caller<15>,
		};
		static const col_caller_t col_callers[MAX_FILTER_RADIUS] =
		{
			0, erose_filter_col_caller<1>, erose_filter_col_caller<2>, erose_filter_col_caller<3>,
			erose_filter_col_caller<4>, erose_filter_col_caller<5>, erose_filter_col_caller<6>,
			erose_filter_col_caller<7>, erose_filter_col_caller<8>, erose_filter_col_caller<9>,
			erose_filter_col_caller<10>, erose_filter_col_caller<11>, erose_filter_col_caller<12>,
			erose_filter_col_caller<13>, erose_filter_col_caller<14>, erose_filter_col_caller<15>,
		};

		if (dim == 0)
		{
			row_callers[radius](dst_d, src_d, nX, nY, pitch);
		}
		if (dim == 1)
		{
			col_callers[radius](dst_d, src_d, nX, nY, pitch);
		}
		cudaSafeCall(hipGetLastError(), "erose_filter");
	}
#pragma endregion

	void DynamicFusionProcessor::eroseColor(const ColorMap& src, ColorMap& dst, int nRadius)
	{
		m_color_tmp.create(src.rows(), src.cols());
		dst.create(src.rows(), src.cols());

		erose_filter((uchar4*)m_color_tmp.ptr(), (const uchar4*)src.ptr(), src.cols(), 
			src.rows(), src.step() / sizeof(uchar4), nRadius, 0);
		erose_filter((uchar4*)dst.ptr(), (const uchar4*)m_color_tmp.ptr(), src.cols(),
			src.rows(), src.step() / sizeof(uchar4), nRadius, 0);
	}
}